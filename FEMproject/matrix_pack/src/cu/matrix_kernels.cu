#include "hip/hip_runtime.h"
#include "matrix_kernels.h"

//hipblasSgemmStridedBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
//                          2, 1, 2,
//                          &alpha,
//                          d_A, 2, 4,
//                          d_B, 2, 2,
//                          &beta,
//                          d_C, 2, 2,
//                          4);


__global__
void show(float* ptr, int size) {
  for(int i =0; i<size; i++)
  printf("%f\n", ptr[i]);
}

float thrust_dotProduct_Ker(float *src1, float *src2, size_t size) {
  return thrust::inner_product(thrust::device,
                               thrust::device_pointer_cast(src1),
                               thrust::device_pointer_cast(src1 + size),
                               thrust::device_pointer_cast(src2), 0.f);
}

void thrust_divideElementwise_Ker(float *src1, float *src2, float *tgt, size_t size) {
  thrust::transform(thrust::device,
                    thrust::device_pointer_cast(src1),
                    thrust::device_pointer_cast(src1 + size),
                    thrust::device_pointer_cast(src2),
                    thrust::device_pointer_cast(tgt),
                    thrust::divides<float>());
}

void thrust_sortByKey_Ker(float *keys, float *src, size_t size) {
  thrust::sort_by_key(thrust::device,
                      thrust::device_pointer_cast(keys),
                      thrust::device_pointer_cast(keys + size),
                      thrust::device_pointer_cast(src));
}

void thrust_reduceByKey_Ker(float *keys, float *src, float *tgt, size_t size) {
  thrust::reduce_by_key(thrust::device,
                        thrust::device_pointer_cast(keys),
                        thrust::device_pointer_cast(keys + size),
                        thrust::device_pointer_cast(src),
                        thrust::make_discard_iterator(),
                        thrust::device_pointer_cast(tgt));
}

void thrust_sort_Ker(float *src, size_t size) {
  try {
    thrust::stable_sort(thrust::device,
                        thrust::device_pointer_cast(src),
                        thrust::device_pointer_cast(src + size));
  } catch(thrust::system_error e) {
    std::cerr << "Error inside sort: " << e.what() << std::endl;
    EXIT_FAILURE;
  }
}

void thrust_setTo_Ker(float *src, size_t size, float v) {
  thrust::fill(thrust::device,
               thrust::device_pointer_cast(src),
               thrust::device_pointer_cast(src + size), v);
}

__global__
void kernelMultiply(const size_t n, const float *data, const float *vec, const size_t size, float *tgt) {
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {
    for (size_t j = 0; j < size; ++j) {
      tgt[j + index * size] = 0.f;
      for (size_t k = 0; k < size; ++k) {
        tgt[j + index * size] += data[k + j * size + index * size * size] * vec[k + index * size];
      }
    }
  }
}

void multiplyByVec_Ker(const size_t numMatr, const float *data,
                       const float *vec_data, const size_t vecSize,
                       float *tgt) {
  kernelMultiply<<<(numMatr + 255) / 256, 256>>>(numMatr, data, vec_data, vecSize, tgt);
}

__global__
void kernelSetTo(const size_t size, float *data, const float value) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < size) {
    data[id] = value;
  }
}

void setTo_Ker(const size_t size, float *data, const float value) {
  kernelSetTo<<<(size + 255) / 256, 256>>>(size, data, value);
}

__global__
void kernelScale(const size_t size, float *data, const float value) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < size) {
    data[id] *= value;
  }
}

void scale_Ker(const size_t size, float *data, const float value) {
  kernelScale<<<(size + 255) / 256, 256>>>(size, data, value);
}

__global__
void kernelAddWeighted(const size_t size, float *data, const float *src,
                       const float alpha, const float beta) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < size) {
    data[id] = alpha * data[id] + beta * src[id];
  }
}

void addWeighted_Ker(const size_t size, float *data, const float *src,
                     const float alpha, const float beta) {
  kernelAddWeighted<<<(size + 255) / 256, 256>>>(size, data, src, alpha, beta);
}

__host__ __device__ unsigned int
IDX(unsigned int i,unsigned  int j,unsigned int ld){
  return j*ld+i;
}

__device__ float
det_kernel(float *a_copy,unsigned int *n,hipblasHandle_t *hdl){
  int *info = (int *)malloc(sizeof(int));info[0]=0;
  int batch=1;int *p = (int *)malloc(*n*sizeof(int));
  float **a = (float **)malloc(sizeof(float *));
  *a = a_copy;
  hipblasStatus_t status=hipblasSgetrfBatched(*hdl, *n, a, *n, p, info, batch);
  hipDeviceSynchronize();
  unsigned int i1;
  float res=1;
  for(i1=0;i1<(*n);++i1)res*=a_copy[IDX(i1,i1,*n)];
  return res;
}
